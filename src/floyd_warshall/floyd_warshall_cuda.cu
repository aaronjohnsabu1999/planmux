#include <hip/hip_runtime.h>
#include <omp.h>
#include <vector>
#include <fstream>
#include <iostream>
#include <cstdlib>
#include <cstdio>

#define	INFTY	1e8

using namespace std;

__global__ void calcH(int V, int mid, int *g, int *h)
{
  int start = threadIdx.y + blockIdx.y * blockDim.y;
  int end   = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (start < V && end < V) {
    h[start*V + end] = min(g[start*V + end], g[start*V + mid] + g[mid*V + end]);
  }
}


__global__ void calcG(int V, int *g, int *h)
{
  int start = threadIdx.y + blockIdx.y * blockDim.y;
  int end   = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (start < V && end < V) {
    g[start*V + end] = h[start*V + end];
  }
}

int main(int argv, char **argc)
{
  // Check if the number of arguments is correct
  if (argv != 2) {
    fprintf(stderr, "Usage: %s <number_of_vertices>\n", argc[0]);
    return 1;
  }
  // Define the number of vertices and edges
  int V = strtol(argc[1], (char **)NULL, 10), E = 0;
  int *g, *h;
  
  // Allocate memory for the graph and the intermediate results
  hipMallocManaged(&g, V*V*sizeof(int));
  hipMallocManaged(&h, V*V*sizeof(int));
  
  // Initialize the graph with random weights
  for (int start = 0; start < V; start++)
    for (int end = 0; end < V; end++)
      if (rand()%((V*V)/(3*V)))
	  {
	    g[start*V + end] = rand()%500 + 1;
        E++;
      }
      else
        g[start*V + end] = INFTY;
  
  // Initialize the intermediate results
  int threads = 10;
  int blocks  = (V + threads - 1) / threads;
  dim3 THREADS (threads, threads);
  dim3 BLOCKS  ( blocks,  blocks);
  
  for (int mid = 0; mid < V; mid++)
  {
    calcH<<<BLOCKS, THREADS>>>(V, mid, g, h);
    hipDeviceSynchronize();
    calcG<<<BLOCKS, THREADS>>>(V, g, h);
    hipDeviceSynchronize();
  }
  
  printf("Time taken for CUDA implementation with (V = %d) = ", V);
  return 0;
}